#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <assert.h>
#include <iostream>
#include <time.h>

using namespace std;

__global__ void kernel_histgram_01(unsigned char* buffer, int size, unsigned int* histo)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    int tid = idx + idy * blockDim.x * gridDim.x;
    if (tid >= size) return;

    unsigned char v = buffer[tid];
    atomicAdd(&histo[v], 1);
}

/*
һ���߳�����32�߳�
ÿ����߳����Ķ������ϲ�����һ�Σ����δ�����С32�ֽڣ����128���ֽ�
128 / 16 = 8��ÿ���̴߳���8���ֽ�Ч�����
*/
__global__ void kernel_histgram_02(unsigned char* buffer, int size, unsigned int* histo)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    int tid = idx + idy * blockDim.x * gridDim.x * 8;
    if (tid >= size) return;

    long long v = *(long long*)(&buffer[tid]);
    atomicAdd(&histo[v & 0x00000000000000FF], 1);
    atomicAdd(&histo[v & 0x000000000000FF00 >> 8], 1);
    atomicAdd(&histo[v & 0x0000000000FF0000 >> 16], 1);
    atomicAdd(&histo[v & 0x00000000FF000000 >> 24], 1);
    atomicAdd(&histo[v & 0x000000FF00000000 >> 32], 1);
    atomicAdd(&histo[v & 0x0000FF0000000000 >> 40], 1);
    atomicAdd(&histo[v & 0x00FF000000000000 >> 48], 1);
    atomicAdd(&histo[v & 0xFF00000000000000 >> 56], 1);
}

/*
ƿ������atomicAdd�����ԭ�Ӳ������������˲��е�������
�����ö�����������ڴ����atomicAdd�����оֲ����ܣ�
�����ջ�ʹ��atomicAdd������atomicAdd�ĵ��ô�����
*/
__shared__ unsigned int d_bin_data_shared[256];
__global__ void kernel_histgram_03(unsigned char* buffer, int size, unsigned int* histo)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    int tid = idx + idy * blockDim.x * gridDim.x * 8;
    if (tid >= size) return;

    int tidInBlock = threadIdx.y * blockDim.x + threadIdx.x;
    if (tidInBlock < 256) {
        d_bin_data_shared[tidInBlock] = 0;
    }
    __syncthreads();

    unsigned char v = buffer[tid];
    ++d_bin_data_shared[v];
    __syncthreads();

    atomicAdd(&histo[tidInBlock], d_bin_data_shared[tidInBlock]);
}

void histgram_cpu(unsigned char* buffer, int size, unsigned int* histo)
{
    clock_t st = clock();
    cout << "caculate with cpu:" << endl;
    for (int i = 0; i < size; i++) {
        auto v = buffer[i];
        ++histo[v];
    }
    cout << "  timespend " << clock() - st << " ms" << endl;
}

int main()
{
    unsigned char *h_hist_data = nullptr;
    unsigned char *d_hist_data = nullptr;

    unsigned int h_bin_data[256];
    unsigned int *d_bin_data = nullptr;

    // ׼������
    cout << "preparing data:" << endl;
    int length = 1E+8;
    h_hist_data = new unsigned char[length];
    memset(h_hist_data, 0x00, length);

    hipMalloc(&d_hist_data, length);
    hipMalloc(&d_bin_data, 256 * sizeof(int));
    hipMemset(d_bin_data, 0, 255 * sizeof(int));

    for (int i = 0; i < length; i++) {
        //auto v = (unsigned char)(rand() % 250 + 2);
        auto v = (unsigned char)(i % 256);
        h_hist_data[i] = v;
    }
    hipMemcpy(d_hist_data, h_hist_data, length, hipMemcpyHostToDevice);
    cout << "  ok" << endl;

    // cpu����
    histgram_cpu(h_hist_data, length, h_bin_data);

    // ��ʱ����
    hipEvent_t ev0, ev1;
    hipEventCreate(&ev0);
    hipEventCreate(&ev1);


    // ���ü���
    cout << "caculate with gpu:"<< endl;
    /*const int threadCount = 256;
    dim3 tn(threadCount);
    dim3 bn(length / threadCount + 1);
    hipEventRecord(ev0);
    kernel_histgram_01 << <bn, tn >> >(d_hist_data, length, d_bin_data);
    hipEventRecord(ev1);
    hipEventSynchronize(ev1);*/

    // �߳����Ż���Ч������
    /*const int threadCount = 256;
    dim3 tn(threadCount);
    dim3 bn(length / 8 / threadCount + 1);
    hipEventRecord(ev0);
    kernel_histgram_02 << <bn, tn >> >(d_hist_data, length, d_bin_data);
    hipEventRecord(ev1);
    hipEventSynchronize(ev1);*/

    const int threadCount = 512;
    dim3 tn(threadCount);
    dim3 bn(length / threadCount + 1);
    hipEventRecord(ev0);
    kernel_histgram_03 << <bn, tn >> >(d_hist_data, length, d_bin_data);
    hipEventRecord(ev1);
    hipEventSynchronize(ev1);

    // ����
    cout << "============analysis===========" << endl;
    hipMemcpy(h_bin_data, d_bin_data, 256 * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < 256; i++) {
        cout << i << " : " <<h_bin_data[i] << endl;
    }


    // ��ʱ�׶�
    float timespend = 0;
    hipEventElapsedTime(&timespend, ev0, ev1);
    cout << "  timespend " << timespend << " ms" << endl;

    cin.ignore();

    return 0;
}